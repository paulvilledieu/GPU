#include "hip/hip_runtime.h"
#include "render.hpp"
#include <cassert>
#include <iostream>

  [[gnu::noinline]]
void _abortError(const char* msg, const char* fname, int line)
{
  hipError_t err = hipGetLastError();
  //spdlog::error("{} ({}, line: {})", msg, fname, line);
  //spdlog::error("Error {}: {}", hipGetErrorName(err), hipGetErrorString(err));
  std::exit(1);
}

#define abortError(msg) _abortError(msg, __FUNCTION__, __LINE__)

// Executed on device, called by host
__global__ void kernel_dilation(unsigned char* dst_device, unsigned char *src_device, int structuring_radius,
    int width, int height)
{
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  int start_x = x-structuring_radius < 0 ? 0 : x-structuring_radius;
  int start_y = y-structuring_radius < 0 ? 0 : y-structuring_radius;
  int end_x = x+structuring_radius > width - 1 ? width - 1 : x+structuring_radius;
  int end_y = y+structuring_radius > height - 1 ? height - 1 : y+structuring_radius;
  bool stop = false;

  for (int i = start_y; i <= end_y && !stop; ++i)
  {
    for (int j = start_x; j <= end_x && !stop; ++j)
    {
      if (src_device[i*width+ j] == 0)
      {
        dst_device[y*width+ x] = 0;
        stop = true;
      }
    }
  }

  if (!stop)
    dst_device[y*width+x] = 255;
}

__global__ void kernel_erosion(unsigned char* dst_device, unsigned char *src_device, int structuring_radius,
    int width, int height)
{
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  int start_x = x-structuring_radius < 0 ? 0 : x-structuring_radius;
  int start_y = y-structuring_radius < 0 ? 0 : y-structuring_radius;
  int end_x = x+structuring_radius > width - 1 ? width - 1 : x+structuring_radius;
  int end_y = y+structuring_radius > height - 1 ? height - 1 : y+structuring_radius;
  bool stop = false;

  for (int i = start_y; i <= end_y && !stop; ++i)
  {
    for (int j = start_x; j <= end_x && !stop; ++j)
    {
      if (src_device[i*width+ j] == 255)
      {
        dst_device[y*width+ x] = 255;
        stop = true;
      }
    }
  }

  if (!stop)
    dst_device[y*width+x] = 0;
}

__global__ void kernel_dilation_sep_1(unsigned char* tmp_device, unsigned char *src_device,
    int structuring_radius, int width, int height)
{
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  int start_x = x-structuring_radius < 0 ? 0 : x-structuring_radius;
  int end_x = x+structuring_radius > width - 1? width - 1: x+structuring_radius;
  bool stop = false;

  for (int i = start_x; i <= end_x && !stop; ++i)
  {
    if (src_device[y*width+ i] == 0)
    {
      tmp_device[y*width+ x] = 0;
      stop = true;
    }
  }

  if (!stop)
    tmp_device[y*width+x] = 255;
}

__global__ void kernel_dilation_sep_2(unsigned char* dst_device, unsigned char *tmp_device,
    int structuring_radius, int width, int height)
{
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  int start_y = y-structuring_radius < 0 ? 0 : y-structuring_radius;
  int end_y = y+structuring_radius > height - 1? height - 1: y+structuring_radius;
  bool stop = false;

  for (int j = start_y; j <= end_y && !stop; ++j)
  {
    if (tmp_device[j*width+ x] == 0)
    {
      dst_device[y*width+ x] = 0;
      stop = true;
    }
  }

  if (!stop)
    dst_device[y*width+x] = 255;
}

__global__ void kernel_erosion_sep_1(unsigned char* tmp_device, unsigned char *src_device,
    int structuring_radius, int width, int height)
{
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  int start_x = x-structuring_radius < 0 ? 0 : x-structuring_radius;
  int end_x = x+structuring_radius > width - 1? width - 1: x+structuring_radius;
  bool stop = false;

  for (int i = start_x; i <= end_x && !stop; ++i)
  {
    if (src_device[y*width+ i] == 255)
    {
      tmp_device[y*width+ x] = 255;
      stop = true;
    }
  }

  if (!stop)
    tmp_device[y*width+x] = 0;
}

__global__ void kernel_erosion_sep_2(unsigned char* dst_device, unsigned char *tmp_device,
    int structuring_radius, int width, int height)
{
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  int start_y = y-structuring_radius < 0 ? 0 : y-structuring_radius;
  int end_y = y+structuring_radius > height - 1? height - 1: y+structuring_radius;
  bool stop = false;

  for (int j = start_y; j <= end_y && !stop; ++j)
  {
    if (tmp_device[j*width+ x] == 255)
    {
      dst_device[y*width+ x] = 255;
      stop = true;
    }
  }

  if (!stop)
    dst_device[y*width+x] = 0;
}
/*
__global__ void kernel_erosion_sep_1_shared(unsigned char* tmp_device, unsigned char *src_device,
    int structuring_radius, int width, int height, int tile_width, int tile_height)
{
  //pixel in the image
  int x = blockIdx.x * tile_width + threadIdx.x;
  int y = blockIdx.y * tile_height + threadIdx.y - structuring_radius;

  if (x >= width || y < 0 || x < 0 || y >= height) {
    return;
  }
  extern __shared__ unsigned char shared[];

  //fill shared memory with image data
  shared[threadIdx.y * blockDim.x + threadIdx.x] = src_device[y * width + x];
  //wait for all threads
  __syncthreads();
  if (y < (blockIdx.y * tile_height) || blockIdx.y >= ((blockIdx.y + 1) * tile_height)) {
    return;
  }

  //starting point on the y axis for the structuring element
  int offset = threadIdx.y - structuring_radius;

  //starting point in shared memory
  unsigned char *start = &shared[offset * blockDim.x + threadIdx.x];

  bool stop = false;

  //iterate on the cols for a 1D convolution along the y axis
  for (int i = 0; i <= 2 * structuring_radius && !stop; i++)
  {
    if (start[i * blockDim.x] == 255)
    {
      tmp_device[y * width + x] = 255;
      stop = true;
    }
  }

  if (!stop)
    tmp_device[y * width + x] = 0;
}

__global__ void kernel_erosion_sep_2_shared(unsigned char* tmp_device, unsigned char *src_device,
    int structuring_radius, int width, int height, int tile_width, int tile_height)
{

  int x = blockIdx.x * tile_width + threadIdx.x - structuring_radius;
  int y = blockIdx.y * tile_height + threadIdx.y;

  if (x >= width || y < 0 || x < 0 || y >= height) {
    return;
  }
  extern __shared__ unsigned char shared[];
  shared[threadIdx.y * blockDim.x + threadIdx.x] = src_device[y * width + x];
  __syncthreads();
  if (x < (blockIdx.x * tile_width) || blockIdx.x >= ((blockIdx.x + 1) * tile_width)) {
    return;
  }

  unsigned char *start = &shared[threadIdx.y * blockDim.x + threadIdx.x - structuring_radius];

  bool stop = false;
  for (int i = 0; i <= 2 * structuring_radius && !stop; i++)
  {
    if (start[i] == 255)
    {
      tmp_device[y * width + x] = 255;
      stop = true;
    }
  }

  if (!stop)
    tmp_device[y * width + x] = 0;
}


__global__ void kernel_dilation_sep_1_shared(unsigned char* tmp_device, unsigned char *src_device,
    int structuring_radius, int width, int height, int tile_width, int tile_height)
{
  int x = blockIdx.x * tile_width + threadIdx.x;
  int y = blockIdx.y * tile_height + threadIdx.y - structuring_radius;

  if (x >= width || y < 0 || x < 0 || y >= height) {
    return;
  }
  extern __shared__ unsigned char shared[];
  shared[threadIdx.y * blockDim.x + threadIdx.x] = src_device[y * width + x];
  __syncthreads();
  if (y < (blockIdx.y * tile_height) || blockIdx.y >= ((blockIdx.y + 1) * tile_height)) {
    return;
  }
  int offset = threadIdx.y - structuring_radius;
  unsigned char *start = &shared[offset * blockDim.x + threadIdx.x];

  bool stop = false;
  for (int i = 0; i <= 2 * structuring_radius && !stop; i++)
  {
    if (start[i * blockDim.x] == 0)
    {
      tmp_device[y * width + x] = 0;
      stop = true;
    }
  }

  if (!stop)
    tmp_device[y * width + x] = 255;
}

__global__ void kernel_dilation_sep_2_shared(unsigned char* tmp_device, unsigned char *src_device,
    int structuring_radius, int width, int height, int tile_width, int tile_height)
{

  int x = blockIdx.x * tile_width + threadIdx.x - structuring_radius;
  int y = blockIdx.y * tile_height + threadIdx.y;

  if (x >= width || y < 0 || x < 0 || y >= height) {
    return;
  }
  extern __shared__ unsigned char shared[];
  shared[threadIdx.y * blockDim.x + threadIdx.x] = src_device[y * width + x];
  __syncthreads();
  if (x < (blockIdx.x * tile_width) || blockIdx.x >= ((blockIdx.x + 1) * tile_width)) {
    return;
  }

  unsigned char *start = &shared[threadIdx.y * blockDim.x + threadIdx.x - structuring_radius];

  bool stop = false;
  for (int i = 0; i <= 2 * structuring_radius && !stop; i++)
  {
    if (start[i] == 0)
    {
      tmp_device[y * width + x] = 0;
      stop = true;
    }
  }

  if (!stop)
    tmp_device[y * width + x] = 255;
}
*/

__global__ void kernel_dilation_sep_1_shared(unsigned char* tmp_device, unsigned char *src_device,
    int structuring_radius, int width, int height, int tile_width, int tile_height)

{
  int x = blockIdx.x * tile_width + threadIdx.x;
  int y = blockIdx.y * tile_height + threadIdx.y;

  if (x >= width || y >= height)
    return;

  unsigned char *block_ptr = src_device + blockIdx.x;
  extern __shared__ unsigned char tile[];

  for (int j = threadIdx.y; j < tile_height; j += blockDim.y)
  {
    if (j >= height)
      continue;
    tile[j] = block_ptr[j*width];
  }

  __syncthreads();
  for (int i = threadIdx.y; i < tile_height; i += blockDim.y)
  {
    int start = i - structuring_radius;
    bool stop = false;

    for (int j = start; j <= start + structuring_radius * 2 && !stop; ++j)
    {
      if (j < 0)
        continue;
      if (j < height && tile[j] == 0)
      {
        stop = true;
        tmp_device[i*width+blockIdx.x] = 0;
      }
    }
    if (!stop)
      tmp_device[i*width+blockIdx.x] = 255;
  }
}

__global__ void kernel_dilation_sep_2_shared(unsigned char* tmp_device, unsigned char *src_device,
    int structuring_radius, int width, int height, int tile_width, int tile_height)

{
  int x = blockIdx.x * tile_width + threadIdx.x;
  int y = blockIdx.y * tile_height + threadIdx.y;

  if (x >= width || y >= height)
    return;

  unsigned char *block_ptr = src_device + blockIdx.y*width;
  extern __shared__ unsigned char tile[];

  for (int j = threadIdx.x; j < tile_width; j += blockDim.x)
  {
    if (j >= width)
      continue;
    tile[j] = block_ptr[j];
  }

  __syncthreads();
  for (int i = threadIdx.x; i < tile_width; i += blockDim.x)
  {
    int start = i - structuring_radius;
    bool stop = false;

    for (int j = start; j <= start + structuring_radius * 2 && !stop; ++j)
    {
      if (j < 0)
        continue;
      if (j < width && tile[j] == 0)
      {
        stop = true;
        tmp_device[blockIdx.y*width+i] = 0;
      }
    }
    if (!stop)
      tmp_device[blockIdx.y*width+i] = 255;
  }
}

void run_kernel(std::string type, unsigned char* dst_device, unsigned char* src_device,
    int width, int height, int structuring_radius, std::ptrdiff_t stride)
{
  int bsize = 2;
  dim3 block(bsize, bsize);

  int size_x = std::ceil((float)width / bsize);
  int size_y = std::ceil((float)height / bsize);
  printf("width %d %d \n", width, height);
  printf("a %d b %d\n", size_x, size_y);
  dim3 grid(size_x, size_y);

  if (!type.compare("dilation"))
    kernel_dilation<<<grid, block>>>(dst_device, src_device, structuring_radius, width, height);
  else if (!type.compare("erosion"))
    kernel_erosion<<<grid, block>>>(dst_device, src_device, structuring_radius, width, height);
  else
  {
    unsigned char *tmp_device;
    hipHostMalloc((void**)&tmp_device, width * height * sizeof(unsigned char *));

    if (!type.compare("dilation_sep"))
    {
      kernel_dilation_sep_1<<<grid, block>>>(tmp_device, src_device, structuring_radius, width, height);
      hipError_t cudaerr = hipDeviceSynchronize();
      kernel_dilation_sep_2<<<grid, block>>>(dst_device, tmp_device, structuring_radius, width, height);

    }
    else if (!type.compare("erosion_sep"))
    {
      kernel_erosion_sep_1<<<grid, block>>>(tmp_device, src_device, structuring_radius, width, height);
      hipError_t cudaerr = hipDeviceSynchronize();
      kernel_erosion_sep_2<<<grid, block>>>(dst_device, tmp_device, structuring_radius, width, height);
    }
    else if (!type.compare("erosion_sep_shared"))
    {
      int tile_w = 1;
      /*int tile_h = height;
      dim3 block_shared_y(tile_w, tile_h);
      size_x = std::ceil((float) width / tile_w);
      size_y = std::ceil((float) width / tile_h);
      dim3 grid_shared_y(size_x, size_y);
      kernel_erosion_sep_1_shared<<<grid_shared_y, block_shared_y, tile_w*tile_h*sizeof(unsigned char)>>>(tmp_device, src_device, structuring_radius, width, height, tile_w, tile_h);
      hipError_t cudaerr = hipDeviceSynchronize();
      tile_w = width;
      tile_h = 1;
      dim3 block_shared_x(tile_w, tile_h);
      size_x = std::ceil((float) width / tile_w);
      size_y = std::ceil((float) width / tile_h);
      dim3 grid_shared_x(size_x, size_y);
      kernel_erosion_sep_2_shared<<<grid_shared_x, block_shared_x, tile_w*tile_h*sizeof(unsigned char)>>>(dst_device, tmp_device, structuring_radius, width, height, tile_w, tile_h);
*/
    }
    else if (!type.compare("dilation_sep_shared"))
    {
      /*int tile_w = 1;
      int tile_h = height;
      dim3 block_shared_y(tile_w, tile_h);
      dim3 grid_shared_y(width, 1);
      kernel_dilation_sep_1_shared<<<grid_shared_y, block_shared_y, tile_w*tile_h*sizeof(unsigned char)>>>(dst_device, src_device, structuring_radius, width, height, tile_w, tile_h);
      hipError_t cudaerr = hipDeviceSynchronize();
*/
      int tile_w = width;
      int tile_h = 1;
      dim3 block_shared_x(tile_w, tile_h);
      dim3 grid_shared_x(1, height);
      kernel_dilation_sep_2_shared<<<grid_shared_x, block_shared_x, tile_w*tile_h*sizeof(unsigned char)>>>(dst_device, src_device, structuring_radius, width, height, tile_w, tile_h);
    }

  }

  hipError_t cudaerr = hipDeviceSynchronize();
  //waits for all kernels to run. If there is an error in the kernel it will show here

}
